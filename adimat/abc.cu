#include <iostream>
#include <string>
#include <fstream>
#include <sstream>
#include <bitset>
#include <cstdlib>
#include <cmath>
#include <algorithm>
#include<iomanip>
#include<string.h>
#include<istream>
#include<limits.h>
#include<hip/hip_runtime.h>
using namespace std;

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
 int width;
 int height;
 float* elements;
} Matrix;
// Thread block size
#define BLOCK_SIZE 64
// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);
// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE

char mystring [9999999];
int main()
{

double** M1;
double** M2;

//char a[20];
char* p;
int i,j;
   FILE * pFile;
   
   int cols1=0;
   int rows1=0;
   int cols2,rows2=0;
   pFile = fopen ("input_large.txt" , "r");
     if ( fgets (mystring , 9999999 , pFile) != NULL )
       for(i=0;i<strlen(mystring);i++)
	   {
		   if(mystring[i]=='\t')
			   cols1++;
	   }
     fclose (pFile);
   //cout<<cols1<<endl;
   pFile = fopen ("input_large.txt" , "r");
   rows1=0;
   while(fgets(mystring,9999999,pFile)!=NULL)
   {
    if(mystring[0]=='\n')
	   break;
   rows1++;
   }
  // cout<<rows1<<endl;
  //  pFile = fopen ("input_large.txt" , "r");
   while(fgets(mystring,9999999,pFile)!=NULL)
   {
    cols2=0;    
	 for(i=0;i<strlen(mystring);i++)
	   {
		   if(mystring[i]=='\t')
			   cols2++;
	   }
	   rows2++;
   }	   
   
   //cout<<cols2<<endl<<rows2;
   cols1++;
   cols2++;

   fclose (pFile);
   
//cout<<"hi"<<endl;   
   cout<<cols1<<endl<<rows1<<endl<<cols2<<endl<<rows2<<endl;
   






M1= (double**) malloc(rows1*sizeof(double*));
//cout<<"hi"<<endl;
for(i=0;i<rows1;i++)
M1[i]=(double*) malloc(cols1*sizeof(double));
//cout<<"hello";

M2= (double**) malloc(rows2*sizeof(double*));
for(i=0;i<rows2;i++)
M2[i]=(double*) malloc(cols2*sizeof(double));
//cout<<"hello";

pFile = fopen ("input_large.txt" , "r");
i=0;
while(fgets(mystring,9999999,pFile)!=NULL)
{
	
	if(mystring[0]=='\n')
		goto abc;
	p=mystring;
	
		for(j=0;j<cols1;j++)
			M1[i][j]=strtod(p,&p);

	i++;
}

abc:
i=0;
while(fgets(mystring,9999999,pFile)!=NULL)
{
	p=mystring;
		
		for(j=0;j<cols2;j++)
			M2[i][j]=strtod(p,&p);

	i++;
}

fclose (pFile);

 // Load A and B to device memory

 Matrix A,B,C;
A.width=cols1;
B.width=cols2;
A.height=rows1;
B.height=rows2;
C.width=A.width+B.width-1;
C.height=A.height+B.height-1;
C.elements=(float*)malloc(C.height*C.width*sizeof(float));
//cout<<"hi";

A.elements=(float*)malloc(A.width*A.height*sizeof(float));
//for(i=0;i<(A.width*A.height);i++)
	//A.elements[i]=1;
	i=0;

	for(int g=0;g<rows1;g++)
{
for(int h=0;h<cols1;h++ )
{
A.elements[i]=M1[g][h];
i++;
}
}	

	B.elements=(float*)malloc(B.width*B.height*sizeof(float));
//for(i=0;i<(B.width*B.height);i++)
	//B.elements[i]=1;
	i=0;
for(int q=0;q<rows2;q++)
{
for(int w=0;w<cols2;w++ )
{
B.elements[i]=M2[q][w];
i++;
}
}
//for(i=0;i<(B.height*B.width);i++)
//cout<<B.elements[i]<<"\t";
//cout<<endl;
 Matrix d_A;
 d_A.width = A.width; d_A.height = A.height;
 size_t size = A.width * A.height * sizeof(float);
 hipMalloc(&d_A.elements, size);
 hipMemcpy(d_A.elements, A.elements, size,
 hipMemcpyHostToDevice);
 Matrix d_B;
 d_B.width = B.width; d_B.height = B.height;
 size = B.width * B.height * sizeof(float);
 hipMalloc(&d_B.elements, size);
 hipMemcpy(d_B.elements, B.elements, size,
 hipMemcpyHostToDevice);
 // Allocate C in device memory
 Matrix d_C;
 d_C.width = C.width; d_C.height = C.height;
 size = C.width * C.height * sizeof(float);
 hipMalloc(&d_C.elements, size);
 // Invoke kernel
dim3 threadsPerBlock(BLOCK_SIZE,BLOCK_SIZE); 
    
	dim3 numBlocks((C.width+BLOCK_SIZE-1)/BLOCK_SIZE,(C.height+BLOCK_SIZE-1)/BLOCK_SIZE);
	//dim3 numBlocks((int)ceil((float)(C.width)/BLOCK_SIZE),(int)ceil((float)(C.height)/BLOCK_SIZE));
 MatMulKernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C);
 // Read C from device memory
 hipMemcpy(C.elements, d_C.elements, size,
 hipMemcpyDeviceToHost);
 // Free device memory
 hipFree(d_A.elements);
 hipFree(d_B.elements);
 hipFree(d_C.elements);

 for(i=0;i<(C.width*C.height);i++)
 {
 //if(.elements[i]!=0)
 cout<<C.elements[i]<<" ";
//else break;
 }
 cout<<endl<<i;
 
 }
// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
 // Each thread computes one element of C
 // by accumulating results into Cvalue
 
 int row = blockIdx.y * blockDim.y + threadIdx.y;
 int col = blockIdx.x * blockDim.x + threadIdx.x;
   
   if(row<C.height && col<C.width)
   {
 for (int e = 0; e < B.height; ++e)
 for (int c=0;c<B.width;++c)
 {
 if((row>=e)&& (col>=c) && (A.height>(row-e)) && (A.width>(col-c)))
 C.elements[row * C.width + col] += A.elements[(row-e) * A.width + col-c]
 * B.elements[e * B.width + c];
 }
 
 }
 
 
 }